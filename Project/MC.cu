#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "Gates.h"
#include <stdio.h>
// Gates_cu struct contains all the computation parameters
#define GRID_SIZE 900
#define MAX_TILE 1024
#define UNROLL 64
#define CASES 768  


__global__ void MC(Gates_cu2** gates, int no_of_gates, int* edges, float* random_numbers, \
	float* random_numbers_width, int* l_count, int* l_start, int current_level, int no_of_iterations, int grids, int x_grid){


	Gates_cu2 tmp;

	int i = threadIdx.x;// +blockIdx.x*blockDim.x;
	int idx = threadIdx.x;
	int b = blockIdx.x;
	float temp_max = -10000;

	int count; int start;

	count = l_count[current_level]; //tex1Dfetch(count_ter, current_level);
	start = l_start[current_level];// tex1Dfetch(start_ter, current_level);


	if (i < count){

		tmp = gates[b][start + i];
		int row = (int)floor((float)(tmp.y / GRID_SIZE));
		int column = (int)floor((float)(tmp.x / GRID_SIZE));
		int random_number = row*x_grid + column;

		if (tmp.is_input)
		{
			temp_max = 0;
		}
		else{

			for (int in = tmp.start_in; in < tmp.start_in + tmp.no_of_in; ++in){
				if (gates[b][edges[in]].output_time_of_gate > temp_max){
					temp_max = gates[b][edges[in]].output_time_of_gate;
				}
			}

			tmp.output_time_of_gate = temp_max + (0.15 / 3)*tmp.delay + tmp.delay*(random_numbers[random_number*no_of_iterations+b]) \
				- (0.08 / 3)*(random_numbers_width[random_number*no_of_iterations + b])*tmp.delay;
		}
		gates[b][start + i] = tmp;

	}

}

extern "C" void cuMC(Gates_cu2* gates, int no_of_gates, int* edges, int no_of_edges, int no_of_iterations, float** rad, float** rad_w, \
	int grids, int x_grid, int* l_count, int* l_start, int max_level){

	hipError_t cudaStatus;


	Gates_cu2* gates_cu1;
	Gates_cu2** gates_cu;
	
	if (CASES == 1){

		hipMallocManaged(&gates_cu1, no_of_gates*sizeof(Gates_cu2));
		for (int j = 0; j < no_of_gates; ++j){
			gates_cu1[j] = gates[j];
		}
	}
	else{

		hipMallocManaged(&gates_cu, CASES * sizeof(Gates_cu2*));
		for (int i = 0; i < CASES; ++i){
			hipMallocManaged(&gates_cu[i], no_of_gates * sizeof(Gates_cu2));
		}


		for (int i = 0; i < CASES; ++i){
			for (int j = 0; j < no_of_gates; ++j){
				gates_cu[i][j] = gates[j];
			}
		}
	}

	int* edges_cu;

	hipMallocManaged(&edges_cu, no_of_edges * sizeof(int));
	for (int i = 0; i < no_of_edges; ++i){
		edges_cu[i] = edges[i];
	}



	float* random_numbers;
	hipMallocManaged(&random_numbers, sizeof(float) * grids * no_of_iterations);
	int count = 0;
	for (int i = 0; i < grids; ++i){
		for (int j = 0; j < no_of_iterations; ++j){
			random_numbers[count++] = rad[i][j];
		}
	}


	float* random_numbers_width;
	hipMallocManaged(&random_numbers_width, sizeof(float) * grids * no_of_iterations);
	count = 0;
	for (int i = 0; i < grids; ++i){
		for (int j = 0; j < no_of_iterations; ++j){
			random_numbers_width[count++] = rad_w[i][j];
		}
	}

	int* c_l_count;
	hipMallocManaged(&c_l_count, sizeof(int) * max_level);
	for (int i = 0; i < max_level; ++i)
		c_l_count[i] = l_count[i];

	int* c_l_start;
	hipMallocManaged(&c_l_start, sizeof(int) * max_level);
	for (int i = 0; i < max_level; ++i)
		c_l_start[i] = l_start[i];



	float time_elapsed = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int current_level = 0;
	while (current_level < max_level){
		MC << <CASES, MAX_TILE >> > (gates_cu, no_of_gates, edges_cu, random_numbers, random_numbers_width,\
			c_l_count, c_l_start, current_level, no_of_iterations, grids，x_grid);
		hipDeviceSynchronize();

		current_level++;
	}

	hipEventRecord(stop, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_elapsed, start, stop);
	printf("GPU MC running：%f(ms)\n", time_elapsed);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "!!!! GPU program execution error in 2: cuda Error=%d,(%s)\n", cudaStatus, hipGetErrorString(cudaStatus));
	}





}