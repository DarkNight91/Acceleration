#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "Gates.h"
#include <stdio.h>
// Gates_cu struct contains all the computation parameters
#define GRID_SIZE 900
#define MAX_TILE 1024
#define UNROLL 64
#define CASES 1  
#ifdef __cplusplus
extern "C" {
#endif
	extern __device__ float pdf_m[36][10] = {
			{ 0.0000, 0.0040, 0.0080, 0.0120, 0.0160, 0.0199, 0.0239, 0.0279, 0.0319, 0.0359 },
			{ 0.0398, 0.0438, 0.0478, 0.0517, 0.0557, 0.0596, 0.0636, 0.0675, 0.0714, 0.0753 },
			{ 0.0793, 0.0832, 0.0871, 0.0910, 0.0948, 0.0987, 0.1026, 0.1064, 0.1103, 0.1141 },
			{ 0.1179, 0.1217, 0.1255, 0.1293, 0.1331, 0.1368, 0.1406, 0.1443, 0.1480, 0.1517 },
			{ 0.1554, 0.1591, 0.1628, 0.1664, 0.1700, 0.1736, 0.1772, 0.1808, 0.1844, 0.1879 },
			{ 0.1915, 0.1950, 0.1985, 0.2019, 0.2054, 0.2088, 0.2123, 0.2157, 0.2190, 0.2224 },
			{ 0.2257, 0.2291, 0.2324, 0.2357, 0.2389, 0.2422, 0.2454, 0.2486, 0.2517, 0.2549 },
			{ 0.2580, 0.2611, 0.2642, 0.2673, 0.2704, 0.2734, 0.2764, 0.2794, 0.2823, 0.2852 },
			{ 0.2881, 0.2910, 0.2939, 0.2967, 0.2995, 0.3023, 0.3051, 0.3078, 0.3106, 0.3133 },
			{ 0.3159, 0.3186, 0.3212, 0.3238, 0.3264, 0.3289, 0.3315, 0.3340, 0.3365, 0.3389 },
			{ 0.3413, 0.3438, 0.3461, 0.3485, 0.3508, 0.3531, 0.3554, 0.3577, 0.3599, 0.3621 },
			{ 0.3643, 0.3665, 0.3686, 0.3708, 0.3729, 0.3749, 0.3770, 0.3790, 0.3810, 0.3830 },
			{ 0.3849, 0.3869, 0.3888, 0.3907, 0.3925, 0.3944, 0.3962, 0.3980, 0.3997, 0.4015 },
			{ 0.4032, 0.4049, 0.4066, 0.4082, 0.4099, 0.4115, 0.4131, 0.4147, 0.4162, 0.4177 },
			{ 0.4192, 0.4207, 0.4222, 0.4236, 0.4251, 0.4265, 0.4279, 0.4292, 0.4306, 0.4319 },
			{ 0.4332, 0.4345, 0.4357, 0.4370, 0.4382, 0.4394, 0.4406, 0.4418, 0.4429, 0.4441 },
			{ 0.4452, 0.4463, 0.4474, 0.4484, 0.4495, 0.4505, 0.4515, 0.4525, 0.4535, 0.4545 },
			{ 0.4554, 0.4564, 0.4573, 0.4582, 0.4591, 0.4599, 0.4608, 0.4616, 0.4625, 0.4633 },
			{ 0.4641, 0.4649, 0.4656, 0.4664, 0.4671, 0.4678, 0.4686, 0.4693, 0.4699, 0.4706 },
			{ 0.4713, 0.4719, 0.4726, 0.4732, 0.4738, 0.4744, 0.4750, 0.4756, 0.4761, 0.4767 },
			{ 0.4772, 0.4778, 0.4783, 0.4788, 0.4793, 0.4798, 0.4803, 0.4808, 0.4812, 0.4817 },
			{ 0.4821, 0.4826, 0.4830, 0.4834, 0.4838, 0.4842, 0.4846, 0.4850, 0.4854, 0.4857 },
			{ 0.4861, 0.4864, 0.4868, 0.4871, 0.4875, 0.4878, 0.4881, 0.4884, 0.4887, 0.4890 },
			{ 0.4893, 0.4896, 0.4898, 0.4901, 0.4904, 0.4906, 0.4909, 0.4911, 0.4913, 0.4916 },
			{ 0.4918, 0.4920, 0.4922, 0.4925, 0.4927, 0.4929, 0.4931, 0.4932, 0.4934, 0.4936 },
			{ 0.4938, 0.4940, 0.4941, 0.4943, 0.4945, 0.4946, 0.4948, 0.4949, 0.4951, 0.4952 },
			{ 0.4953, 0.4955, 0.4956, 0.4957, 0.4959, 0.4960, 0.4961, 0.4962, 0.4963, 0.4964 },
			{ 0.4965, 0.4966, 0.4967, 0.4968, 0.4969, 0.4970, 0.4971, 0.4972, 0.4973, 0.4974 },
			{ 0.4974, 0.4975, 0.4976, 0.4977, 0.4977, 0.4978, 0.4979, 0.4979, 0.4980, 0.4981 },
			{ 0.4981, 0.4982, 0.4982, 0.4983, 0.4984, 0.4984, 0.4985, 0.4985, 0.4986, 0.4986 },
			{ 0.4987, 0.4987, 0.4987, 0.4988, 0.4988, 0.4989, 0.4989, 0.4989, 0.4990, 0.4990 },
			{ 0.4990, 0.4991, 0.4991, 0.4991, 0.4992, 0.4992, 0.4992, 0.4992, 0.4993, 0.4993 },
			{ 0.4993, 0.4993, 0.4994, 0.4994, 0.4994, 0.4994, 0.4994, 0.4995, 0.4995, 0.4995 },
			{ 0.4995, 0.4995, 0.4995, 0.4996, 0.4996, 0.4996, 0.4996, 0.4996, 0.4996, 0.4997 },
			{ 0.4997, 0.4997, 0.4997, 0.4997, 0.4997, 0.4997, 0.4997, 0.4997, 0.4997, 0.4998 },
			{ 0.4998, 0.4998, 0.4998, 0.4998, 0.4998, 0.4998, 0.4998, 0.4998, 0.4998, 0.4998 },
	};

	extern __device__ float row[36] = { 0.0, 0.1, 0.2, 0.3, 0.4, 0.5, 0.6, 0.7, 0.8, 0.9, 1.0, 1.1, 1.2, 1.3, 1.4, 1.5, 1.6, 1.7, 1.8, 1.9, 2.0, 2.1, 2.2, 2.3, 2.4, 2.5, 2.6, 2.7, 2.8, 2.9, 3.0, 3.1, 3.2, 3.3, 3.4, 3.5 };
	extern __device__ float col[10] = { 0.00, 0.01, 0.02, 0.03, 0.04, 0.05, 0.06, 0.07, 0.08, 0.09 };
#ifdef __cplusplus
}
#endif


texture<int, 1, hipReadModeElementType> count_ter;
texture<int, 1, hipReadModeElementType> start_ter;
//texture<int, 1, hipReadModeElementType> edges_ter;
//texture<int, 1, hipReadModeElementType> sort_ter;
texture<float, 1, hipReadModeElementType> k_m_ter;

typedef struct {
	float mu;
	float sigma;
}mu_sigma_struct;

__device__ float integrate1(float beta){

	bool positive;
	float b = abs(beta);
	float result;
	int flag = 0;

	int i, j;
	for (i = 0; i < 36; ++i){
		for (j = 0; j < 10; ++j){
			if (b >(row[i] + col[j]))
				continue;
			else{
				flag = 1;
				break;
			}
		}
		if (flag)
			break;
	}

	if ((i - 1) < 0 || (j - 1) < 0)
		result = pdf_m[i][j];
	else if (i > 35 || j > 35)
		result = pdf_m[i - 1][j - 1];
	else
		result = 0.5*(pdf_m[i][j] + pdf_m[i - 1][j - 1]);


	if (beta >= 0){
		return result + 0.5;
	}
	else{
		return 0.5 - result;
	}
}

__device__ float integrate(float beta){

	float step = 0.01f;
	float down_idx = -5;
	int up_idx = (int)(beta - down_idx) / step;
	float micro = down_idx;

	float sum = 0.0f;
	float tmp;

	for (int ii = 0; ii < up_idx; ii++){
		tmp = pow(micro, 2) / 2;
		sum += pow(2.718281828f, -tmp) * step;
		micro += step;
	}

	return sum / sqrt(2 * 3.141592654f);

}


__device__ void sum_function(mu_sigma_struct max_strc, Gates_cu* currentG, float* k_para_matrix, int no_of_pc, \
	int max_k_idx){

	currentG->delay_mu = currentG->gate_mu + max_strc.mu;
	int offset = 2 * no_of_pc;
	int g_id_tmp = currentG->id;
	float tmp = 0.0f;
	float tmp2 = 0.0f;
	float sigma = 0.0f;

	for (int i = 0; i < offset; i++)
	{
		//tmp = tex1Dfetch(k_m_ter, (g_id_tmp * 2 * no_of_pc + i)) + tex1Dfetch(k_m_ter, (max_k_idx * 2 * no_of_pc + i));
		tmp = k_para_matrix[g_id_tmp * 2 * no_of_pc + i] + k_para_matrix[max_k_idx * 2 * no_of_pc + i];
		//tmp =  l_k_m[idx * 40 + i] + l_k_m[idx * 40 + i];
		sigma += pow(tmp, 2);
	}

	currentG->delay_sigma = sqrt(sigma);

}


__device__ void max_function(Gates_cu* gates_t, Gates_cu* currentG, int* edges_t, int no_of_pc, float* k_para_matrix,  \
	int bidx, int current_level){

	int e_idx = edges_t[currentG->start_in];
	float mu_1, mu_2, sigma_1, sigma_2;
	int offset = 2 * no_of_pc;
	int max_k_idx;
	mu_sigma_struct max_strc;
	int current_k_idx;
	max_strc.mu = gates_t[e_idx].delay_mu;
	max_strc.sigma = gates_t[e_idx].delay_sigma;
	max_k_idx = gates_t[e_idx].id;

	for (int i = currentG->start_in + 1; i < currentG->start_in + currentG->no_of_in; i++){
		
		e_idx = edges_t[i];
		mu_1 = max_strc.mu;
		sigma_1 = max_strc.sigma;
		mu_2 = gates_t[e_idx].delay_mu;
		sigma_2 = gates_t[e_idx].delay_sigma;
		current_k_idx = gates_t[e_idx].id;

		
		if (mu_1 - 3 * sigma_1 > mu_2 + 3 * sigma_2)
		{
			continue;
		}

		if (mu_1 + 3 * sigma_1 < mu_2 - 3 * sigma_2)
		{
			max_strc.mu = mu_2;
			max_strc.sigma = sigma_2;
			max_k_idx = current_k_idx;
			continue;
		}
		
		//step 2
		float co_variance = 0.0f;
		float correlation = 0.0f;

		for (int j = 0; j < offset; j++){
			//co_variance += tex1Dfetch(k_m_ter, (max_k_idx * offset + j)) * tex1Dfetch(k_m_ter, (current_k_idx * offset + j));
			co_variance += k_para_matrix[max_k_idx * offset + j] * k_para_matrix[current_k_idx * offset + j];
			//co_variance += k_l[fake + j] * k_l[fake + j];
		}

		correlation = co_variance / (sigma_1 * sigma_2);
		if (correlation > 0.99 && abs(sigma_1 - sigma_2) < 0.1){
			if (mu_1 > mu_2){
				continue;
			}
			else{
				max_strc.mu = mu_2;
				max_strc.sigma = sigma_2;
				max_k_idx = current_k_idx;
				continue;
			}

		}

		//step 3
		
		float alpha = sqrt(abs(pow(sigma_1, 2) + pow(sigma_2, 2) - 2 * co_variance));
		float beta = (mu_1 - mu_2) / alpha;

		float phi = pow(2.718281828f, -beta*beta / 2) / sqrt(2 * 3.141592654f);
		float phi_intg = integrate1(beta);
		float phi_intg_m = integrate1(-beta);

		float sigma_3, mu_3;

		mu_3 = mu_1 * phi_intg + mu_2 * phi_intg_m + alpha * phi;
		float sigma_tmp = (pow(mu_1, 2) + pow(sigma_1, 2)) * phi_intg + (pow(mu_2, 2) + pow(sigma_2, 2)) * phi_intg_m + (mu_1 + mu_2) * alpha * phi - mu_3*mu_3;
		sigma_3 = sqrt(abs(sigma_tmp));
		/*
		step 4
		float S0 = 0.0f;
		for (int j = 0; j < offset; j++){
		float r_1 = tex2D(k_m_ter, max_k_idx, j);
		float r_2 = tex2D(k_m_ter, current_k_idx, j);
		float ar_tmp = (sigma_1*r_1*phi_intg + sigma_2*r_2*phi_intg_m) / sigma_3;
		max_k[j] = ar_tmp;
		S0 += ar_tmp * ar_tmp;
		}


		for (int j = 0; j < offset; j++){
		max_k[j] = max_k[j] * sigma_3 / sqrt(abs(S0));
		}
		*/
		max_strc.mu = mu_3;
		max_strc.sigma = sigma_3;
		

	}

	sum_function(max_strc, currentG, k_para_matrix, no_of_pc, max_k_idx);


}


__global__ void PCA(Gates_cu* gates, int no_of_gates, int* sort, int* edges, float* k_param, \
	int* l_count, int* l_start, int current_level, int no_of_pc, int max_level){

	Gates_cu tmp;
	//current_level=0;
	int i = threadIdx.x;// +blockIdx.x*blockDim.x;
	int idx = threadIdx.x;
	int b = blockIdx.x;

	//while (current_level < max_level){

		int count; int start;

		count = l_count[current_level];
		start = l_start[current_level];


		if (i < count){

			tmp = gates[start + i];

			if (tmp.is_input)
			{
				tmp.delay_mu = tmp.gate_mu;
				tmp.delay_sigma = tmp.gate_sigma;
			}
			else{
				//--max--//
				max_function(gates, &tmp, edges, no_of_pc, k_param, b, current_level); //no change supposed to be on gates[i]
				//printf("%f\n", l_gates[idx].delay_sigma);
				//--sum--//
				//sum_function(max_strc, &l_gates[i], k_param, no_of_pc, gates, edges); // gates[i] param should changes
			}
			//l_gates[tmp.id%MAX_TILE] = tmp;
			gates[start + i] = tmp;
			//test(gates, &l_gates[i], edges);
			//gates[b][sort[start + i]] = l_gates[idx];
			//gates[sort[start + i]] = tmp;
		}
		//current_level++;
		//__syncthreads();
	//}
}

__global__ void timing(Gates_cu* gates, int no_of_pc, float* eigen_v, float** eigen_vec, int x_grids, int no_of_gates)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	if (k < no_of_gates){
		float sigma_of_delay = 0.0f;
		gates[k].gate_mu = 0.0f;  // This is to init mu
		gates[k].gate_sigma = 0.0f; // This is to init sigma
		gates[k].gate_mu = gates[k].delay;
		int row = (int)floor((float)(gates[k].y / GRID_SIZE));
		int column = (int)floor((float)(gates[k].x / GRID_SIZE));

		int i_of_j = row*x_grids + column;

		float k_tmp = 0.0f;


		for (int j = 0; j < no_of_pc; j++)
		{
			if (eigen_v[j] < 0){
				eigen_v[j] = 0;
			}

			//L = sqr eg_v * eg_vec * sigma, the dRdL is the constain for a specific size of gate, i'll update it
			k_tmp = (0.15f / 3)* gates[k].delay * sqrt(eigen_v[j]) * eigen_vec[i_of_j][j];// *sigma_of_L;
			sigma_of_delay += k_tmp * k_tmp;
			//gates[k].k_param[j] = k_tmp;

		}


		for (int j = 0; j < no_of_pc; j++)
		{
			//W = sqr eg_v * eg_vec * sigma, the dRdW is the constain for a specific size of gate, i'll update it
			k_tmp = -(0.08f / 3)* gates[k].delay * sqrt(eigen_v[j]) * eigen_vec[i_of_j][j];// *sigma_of_W;
			sigma_of_delay += k_tmp * k_tmp;
			//gates[k].k_param[j + no_of_pc] = k_tmp;
		}

		/*----get gate[i]'s sigma----*/

		gates[k].gate_sigma = sqrt(sigma_of_delay);
	}
}

extern "C" Gates_cu* cuSSTA(Gates_cu* gates, int no_of_gates, int* sort, int* edges, int no_of_edges, int no_of_pc, float *eigen_values, \
	float **eigen_vectors, int x_grid, float* k_param, int* l_count, int* l_start, int max_level, Gates* gates_t){

	hipError_t cudaStatus;

	int* c_sort;
	hipMallocManaged(&c_sort, sizeof(int) * no_of_gates);
	for (int i = 0; i < no_of_gates; ++i){
		c_sort[i] = sort[i];
	}


	float* k_parameters;
	hipMallocManaged(&k_parameters, sizeof(float) * no_of_gates * no_of_pc * 2);
	for (int i = 0; i < no_of_gates * no_of_pc * 2; ++i){
		k_parameters[i] = k_param[i];
	}


	int* c_l_count;
	hipMallocManaged(&c_l_count, sizeof(int) * max_level);
	for (int i = 0; i < max_level; ++i)
		c_l_count[i] = l_count[i];

	int* c_l_start;
	hipMallocManaged(&c_l_start, sizeof(int) * max_level);
	for (int i = 0; i < max_level; ++i)
		c_l_start[i] = l_start[i];

	Gates_cu* gates_cu1;
	Gates_cu** gates_cu;
	Gates_cu* g_str;
	if (CASES == 1){

		hipMallocManaged(&gates_cu1, no_of_gates*sizeof(Gates_cu));
		hipMallocManaged(&g_str, no_of_gates*sizeof(Gates_cu));
		for (int j = 0; j < no_of_gates; ++j){
			gates_cu1[j] = gates[j];
		}
	}
	else{

		hipMallocManaged(&gates_cu, CASES * sizeof(Gates_cu*));
		for (int i = 0; i < CASES; ++i){
			hipMallocManaged(&gates_cu[i], no_of_gates * sizeof(Gates_cu));
		}


		for (int i = 0; i < CASES; ++i){
			for (int j = 0; j < no_of_gates; ++j){
				gates_cu[i][j] = gates[j];
			}
		}
	}

	float* eigen_v;
	float** eigen_vec;
	int* edges_cu;

	hipMallocManaged(&edges_cu, no_of_edges * sizeof(int));
	hipMallocManaged(&eigen_v, no_of_pc * sizeof(float));
	hipMallocManaged(&eigen_vec, no_of_pc * sizeof(float*));

	for (int i = 0; i < no_of_edges; ++i){
		edges_cu[i] = edges[i];
	}
	for (int i = 0; i < no_of_pc; ++i){
		eigen_v[i] = eigen_values[i];
	}
	for (int i = 0; i < no_of_pc; ++i){
		hipMallocManaged(&eigen_vec[i], no_of_pc * sizeof(float));
		for (int j = 0; j < no_of_pc; ++j){
			eigen_vec[i][j] = eigen_vectors[i][j];
		}
	}

	for (int i = 0; i < no_of_pc; ++i){
		assert(eigen_v[i] == eigen_values[i]);
		for (int j = 0; j < no_of_pc; ++j)
			assert(eigen_vectors[i][j] == eigen_vec[i][j]);
	}



	/////////////////////////////////////////////////////////////////////////////////////////
	dim3 blockn = (no_of_gates - 1) / 1024 + 1;
	dim3 threadn = 1024;

	//hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	if (CASES == 1){
		timing << < blockn, threadn >> > (gates_cu1, no_of_pc, eigen_v, eigen_vec, x_grid, no_of_gates);
		hipDeviceSynchronize();
	}
	else{
		for (int i = 0; i < CASES; ++i){
			timing << < blockn, threadn >> > (gates_cu[i], no_of_pc, eigen_v, eigen_vec, x_grid, no_of_gates);
			hipDeviceSynchronize();
		}
	}


	float time_elapsed = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int current_level = 0;
	while (current_level < max_level){
		PCA << <CASES, MAX_TILE >> > (gates_cu1, no_of_gates, c_sort, edges_cu, k_parameters, \
			c_l_count, c_l_start, current_level, no_of_pc, max_level);
		hipDeviceSynchronize();
		current_level++;
	}

	hipEventRecord(stop, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_elapsed, start, stop);
	printf("GPU running：%f(ms)\n", time_elapsed);


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "!!!! GPU program execution error in 2: cuda Error=%d,(%s)\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	return gates_cu1;

}
